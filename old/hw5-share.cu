#include "hip/hip_runtime.h"
%%cuda
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define THREADS_PER_BLOCK 16  // CUDA Kernel block size

double gettime(void) {
    struct timeval tval;
    gettimeofday(&tval, NULL);
    return (double)tval.tv_sec + (double)tval.tv_usec / 1000000.0;
}

__device__ int getNeighborCount(int *grid, int x, int y, int cols, int rows) {
    int count = 0;
    for (int dx = -1; dx <= 1; dx++) {
        for (int dy = -1; dy <= 1; dy++) {
            if (dx == 0 && dy == 0) continue;  // Skip the current cell
            count += grid[(y + dy) * cols + (x + dx)];
        }
    }
    return count;
}

__global__ void nextGeneration(int *current, int *next, int rows, int cols, int *change_flag) {
    // Calculate global indices
    int x = blockIdx.x * blockDim.x + threadIdx.x + 1;  // Offset by 1 for ghost cells
    int y = blockIdx.y * blockDim.y + threadIdx.y + 1;

    // Shared memory allocation (includes halo cells)
    __shared__ int shared_grid[THREADS_PER_BLOCK + 2][THREADS_PER_BLOCK + 2];

    // Local thread indices within the block
    int local_x = threadIdx.x + 1;  // Shared memory index (offset for halo)
    int local_y = threadIdx.y + 1;

    // Load data into shared memory (including halos)
    shared_grid[local_y][local_x] = current[y * (cols + 2) + x];

    // Load halo cells
    if (threadIdx.x == 0) {
        shared_grid[local_y][0] = current[y * (cols + 2) + (x - 1)];  // Left halo
    }
    if (threadIdx.x == blockDim.x - 1) {
        shared_grid[local_y][local_x + 1] = current[y * (cols + 2) + (x + 1)];  // Right halo
    }
    if (threadIdx.y == 0) {
        shared_grid[0][local_x] = current[(y - 1) * (cols + 2) + x];  // Top halo
    }
    if (threadIdx.y == blockDim.y - 1) {
        shared_grid[local_y + 1][local_x] = current[(y + 1) * (cols + 2) + x];  // Bottom halo
    }

    // Load corner halos
    if (threadIdx.x == 0 && threadIdx.y == 0) {
        shared_grid[0][0] = current[(y - 1) * (cols + 2) + (x - 1)];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == 0) {
        shared_grid[0][local_x + 1] = current[(y - 1) * (cols + 2) + (x + 1)];
    }
    if (threadIdx.x == 0 && threadIdx.y == blockDim.y - 1) {
        shared_grid[local_y + 1][0] = current[(y + 1) * (cols + 2) + (x - 1)];
    }
    if (threadIdx.x == blockDim.x - 1 && threadIdx.y == blockDim.y - 1) {
        shared_grid[local_y + 1][local_x + 1] = current[(y + 1) * (cols + 2) + (x + 1)];
    }

    __syncthreads();  // Ensure all threads have loaded their data

    if (x > cols || y > rows) return;  // Bounds check (excluding ghost cells)

    // Calculate the number of live neighbors from shared memory
    int neighbors = 0;
    for (int dy = -1; dy <= 1; dy++) {
        for (int dx = -1; dx <= 1; dx++) {
            if (dx == 0 && dy == 0) continue;
            neighbors += shared_grid[local_y + dy][local_x + dx];
        }
    }

    int idx = y * (cols + 2) + x;
    int new_state = shared_grid[local_y][local_x];

    // Apply Game of Life rules
    if (new_state == 1 && (neighbors < 2 || neighbors > 3)) {
        new_state = 0;  // Cell dies
    } else if (new_state == 0 && neighbors == 3) {
        new_state = 1;  // Cell becomes alive
    }

    // Update flag if there's a change
    if (new_state != current[idx]) {
        *change_flag = 1;
    }
    
    next[idx] = new_state;
}

void initializeGrid(int *grid, int rows, int cols) {
    for (int i = 1; i <= rows; i++) {
        for (int j = 1; j <= cols; j++) {
            grid[i * (cols + 2) + j] = rand() % 2;  // Initialize inner grid randomly
        }
    }
}

void updateGhostCells(int *grid, int rows, int cols) {
    // Copy edges to ghost cells
    for (int i = 1; i <= rows; i++) {
        grid[i * (cols + 2)] = grid[i * (cols + 2) + cols];           // Left ghost column
        grid[i * (cols + 2) + cols + 1] = grid[i * (cols + 2) + 1];   // Right ghost column
    }
    for (int j = 1; j <= cols; j++) {
        grid[j] = grid[rows * (cols + 2) + j];                        // Top ghost row
        grid[(rows + 1) * (cols + 2) + j] = grid[(1) * (cols + 2) + j]; // Bottom ghost row
    }

    // Corners
    grid[0] = grid[rows * (cols + 2)];                                // Top-left corner
    grid[(cols + 1)] = grid[rows * (cols + 2) + cols];                // Top-right corner
    grid[(rows + 1) * (cols + 2)] = grid[(1) * (cols + 2)];           // Bottom-left corner
    grid[(rows + 1) * (cols + 2) + (cols + 1)] = grid[(1) * (cols + 2) + cols];  // Bottom-right corner
}

int main() {
    double starttime = gettime();
    srand(100);

    int rows = 50;
    int cols = 50;
    int iterations = 100;

    int *h_current, *h_next;
    int *d_current, *d_next;
    int *d_change_flag, h_change_flag;
    size_t size = (rows + 2) * (cols + 2) * sizeof(int);  // Include ghost cells

    // Allocate host memory
    h_current = (int *)malloc(size);
    h_next = (int *)malloc(size);

    // Initialize the grid randomly
    initializeGrid(h_current, rows, cols);
    updateGhostCells(h_current, rows, cols);

    // Allocate device memory
    hipMalloc((void **)&d_current, size);
    hipMalloc((void **)&d_next, size);
    hipMalloc((void **)&d_change_flag, sizeof(int));

    // Copy the initial grid to the device
    hipMemcpy(d_current, h_current, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(THREADS_PER_BLOCK, THREADS_PER_BLOCK);
    dim3 numBlocks((cols + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, (rows + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK);
    
    int lastI;
    for (int i = 0; i < iterations; i++) {
        h_change_flag = 0;
        hipMemcpy(d_change_flag, &h_change_flag, sizeof(int), hipMemcpyHostToDevice);
        
        nextGeneration<<<numBlocks, threadsPerBlock>>>(d_current, d_next, rows, cols, d_change_flag);
        hipMemcpy(&h_change_flag, d_change_flag, sizeof(int), hipMemcpyDeviceToHost);
        
        // Break if no changes occurred
        if (h_change_flag == 0) {
            lastI = i;
            break;
        }
        
        hipMemcpy(d_current, d_next, size, hipMemcpyDeviceToDevice);
        lastI = i;
    }

    // Copy the final grid back to the host
    hipMemcpy(h_current, d_current, size, hipMemcpyDeviceToHost);

    double endtime = gettime();

    // Save final board to file
    FILE *outFile = fopen("output.txt", "w");
    if (outFile == NULL) {
        fprintf(stderr, "Error opening file for writing.\n");
        return -1;
    }
    fprintf(outFile, "======\nGEN %d\n======\n", lastI + 1);
    for (int y = 1; y <= rows; y++) {
        for (int x = 1; x <= cols; x++) {
            fprintf(outFile, "%d ", h_current[y * (cols + 2) + x]);
        }
        fprintf(outFile, "\n");
    }
    fprintf(outFile, "\nTime taken = %lf seconds\n", endtime - starttime);
    fclose(outFile);

    // free memory
    hipFree(d_current);
    hipFree(d_next);
    hipFree(d_change_flag);
    free(h_current);
    free(h_next);

    printf("Program ran successfully. Final matrix saved to output.txt.\n");
    return 0;
}
